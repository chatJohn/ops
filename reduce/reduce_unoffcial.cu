
#include <hip/hip_runtime.h>


__global__ void reduce0(float *d_A, const int N){
    extern __shared__ float data[];
    // 从global memory读取数据到shared memory
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    data[tid] = index < N ? d_A[index] : 0.f;
    __syncthreads();

    for(int s = 1; s < blockDim.x; s <<= 1){
        if((tid % (s * 2)) == 0){
            data[tid] += data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        d_A[blockIdx.x] = data[0];
    }
}

__global__ void reduce1(float *d_A, const int N){
    extern __shared__ float data[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[tid] = idx < N ? d_A[idx] : 0.f;
    __syncthreads();

    for(int s = 1; s < blockDim.x; s <<= 1){
        int index = tid * s * 2;
        if(index < blockDim.x){
            data[index] += data[index + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        d_A[blockIdx.x] = data[0];
    }
}

__global__ void reduce2(float *d_A, const int N){
    extern __shared__ float data[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[tid] = idx < N ? d_A[idx] : 0.f;
    __syncthreads();

    for(int s = blockDim.x >> 1; s > 0; s >>= 1){
        if(tid < s){
            data[tid] += data[tid + s];
        } 
        __syncthreads();
    }
    if(tid == 0){
        d_A[blockIdx.x] = data[0];
    }
}

// 在读取的时候就进行规约
__global__ void reduce3(float *d_A, const int N){
    extern __shared__ float data[];
    int tid = threadIdx.x;
    int idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    float sum = idx < N ? d_A[idx] : 0.f;
    if(idx + blockDim.x < N){
        data[tid] = sum + d_A[idx + blockDim.x];
    }
    __syncthreads();

    for(int s = blockDim.x >> 1; s > 0; s >>= 1){
        if(tid < s){
            data[tid] += data[tid + s];
        } 
        __syncthreads();
    }
    if(tid == 0){
        d_A[blockIdx.x] = data[0];
    }
}

#define WRAP_SIZE 32
template<int kWarpSize = WRAP_SIZE>
__device__ __forceinline__ float wrapReduce(float val){
    #pragma unroll
    for(int mask = kWarpSize >> 1; mask >= 1; mask >>= 1){
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

__global__ void reduce4(float *d_A, const int N){
    extern __shared__ float data[];
    int tid = threadIdx.x;
    int idx = 2 * blockIdx.x * blockDim.x + tid;
    float sum = idx < N ? d_A[idx] : 0.f;
    if(idx + blockDim.x < N){
        data[tid] = sum + d_A[idx + blockDim.x];
    }
    __syncthreads();

    for(int s = blockDim.x >> 1; s >= 32; s >>= 1){
        if(tid < s){
            data[tid] = sum = sum + data[tid + s];
        } 
        __syncthreads();
    }

    // warp reduction for last 32 threads
    if(tid < 32){
        sum = wrapReduce<WRAP_SIZE>(sum);
    }

    if(tid == 0){
        d_A[blockIdx.x] = sum;
    }
}